#include "hip/hip_runtime.h"
#include "cusumlib.h"
#include "hip/hip_runtime.h"
#include <random>
#include "hip/hip_runtime_api.h"

__global__ void vectorAdd(float* a, float* b, float* c){
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    /*поскольку это ядро будет выполнятся сразу на множестве
    нитей, каждая из которых сложит свои элементы, нам надо
    узнать индекс текущей нити, чтобы сложить правильные элементы*/
    c[index] = a[index] + b[index];

}

void cusum(int blockSize, int numBlocks, float* a, float* b, float* c){
	hipSetDevice(0);

	int numItems = blockSize * numBlocks;

	float *adev, *bdev, *cdev;

    hipMalloc((void**)&adev, numItems * sizeof(float));

	hipMalloc((void**)&bdev, numItems * sizeof(float));

	hipMalloc((void**)&cdev, numItems * sizeof(float));

    hipMemcpy(adev, a, numItems * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(bdev, b, numItems * sizeof(float), hipMemcpyHostToDevice);

    vectorAdd<<<numBlocks, blockSize>>>(adev, bdev, cdev);

    hipMemcpy((void *) c, cdev, numItems * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(adev);
    hipFree(bdev);
    hipFree(cdev);
}

void randomInit(float* a, int n){
    for (int f = 0; f < n; f++){
        a[f] = rand() / (float) RAND_MAX;
    }
}
